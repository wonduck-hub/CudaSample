#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kernel.cuh"
#include <ctime>

int main(void) {
	hipSetDevice(0);

	int matrix0_col = 300;
	int n = 100;
	int matrix1_row = 200;

	double* matrix0 = (double*)malloc(sizeof(double) * matrix0_col * n);
	double* matrix1 = (double*)malloc(sizeof(double) * n * matrix1_row);
	double* c_output = (double*)malloc(sizeof(double) * matrix0_col * matrix1_row);

	double* d_matrix0;
	double* d_matrix1;
	double* d_c_output;

	hipMalloc(&d_matrix0, sizeof(double) * matrix0_col * n);
	hipMalloc(&d_matrix1, sizeof(double) * n * matrix1_row);
	hipMalloc(&d_c_output, sizeof(double) * matrix0_col * matrix1_row);

	for (int i = 0; i < matrix0_col * n; ++i) {
		matrix0[i] = 1.0;
	}

	for (int i = 0; i < n * matrix1_row; ++i) {
		matrix1[i] = 1.0;
	}

	for (int i = 0; i < matrix0_col * matrix1_row; ++i) {
		c_output[i] = 0.0;
	}

	hipMemcpy(d_matrix0, matrix0, sizeof(double) * matrix0_col * n, hipMemcpyHostToDevice);
	hipMemcpy(d_matrix1, matrix1, sizeof(double) * n * matrix1_row, hipMemcpyHostToDevice);
	hipMemcpy(d_c_output, c_output, sizeof(double) * matrix0_col * matrix1_row, hipMemcpyHostToDevice);

	hipDeviceProp_t prop; // ����̽��� ������Ƽ�� ������ ����ü
	hipGetDeviceProperties(&prop, 0); // 0�� ����̽��� ������ ����ü�� ����
	int n_threads = prop.maxThreadsPerBlock; // ��ϴ� �ִ� ������ ����. �ִ� �� ���� ����� �� �ִ��� Ȯ�ο�
	// �ִ�� ����ϳ� ���� �ھ� ���� ����ϵ� ���̰� ���� ����.

	int n_blocks = prop.multiProcessorCount; // ��Ƽ ���μ����� ����. ��Ƽ ���μ����� ����� �ϳ��� �Ҵ��Ѵ�.

	// GPU computing
	hipEvent_t d_start, d_stop;

	hipEventCreate(&d_start);
	hipEventCreate(&d_stop);

	hipEventRecord(d_start);

	// <<<B, T>>> B�� ����� ����, T�� ��ϴ� �������� ����
	kernel <<<n_blocks, n_threads >>> (matrix0_col, n, matrix1_row, d_matrix0, d_matrix1, d_c_output);

	hipEventRecord(d_stop);

	hipMemcpy(c_output, d_c_output, sizeof(double) * matrix0_col * matrix1_row, hipMemcpyDeviceToHost);

	hipEventSynchronize(d_stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, d_start, d_stop);

	printf("GPU time: %f ms\n", elapsedTime);

	// CPU computing
	double* cpu_c_output = (double*)malloc(sizeof(double) * matrix0_col * matrix1_row);
	
	for (int i = 0; i < matrix0_col * matrix1_row; ++i) {
		cpu_c_output[i] = 0.0;
	}
	
	clock_t start = clock();
	clock_t diff;

	for (int i = 0; i < matrix0_col * matrix1_row; ++i) {
		get_multiplied_matrix(matrix0_col, n, matrix1_row, i, matrix0, matrix1, cpu_c_output);
	}

	diff = clock() - start;
	int msec = diff * 1000 / CLOCKS_PER_SEC;

	printf("CPU time: %d ms\n", msec);

	// print result
	for (int i = 0; i < matrix1_row; ++i) {
		for (int j = 0; j < matrix0_col; ++j) {
			printf("%f ", c_output[i * matrix0_col + j]);
		}
		printf("\n");
	}

	hipFree(d_matrix0);
	hipFree(d_matrix1);
	hipFree(d_c_output);

	free(matrix0);
	free(matrix1);
	free(c_output);
	free(cpu_c_output);
}